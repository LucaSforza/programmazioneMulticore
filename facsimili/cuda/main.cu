#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void run(void) {
    printf("Hello World!\n");
}

int main(void) {
    run<<<1,10>>>();
    hipDeviceSynchronize();
    return 1;
}